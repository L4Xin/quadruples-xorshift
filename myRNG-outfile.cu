#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <bitset>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <chrono>
#include <random>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)
#define N 128
using namespace std;

__global__ void my_generateRandom(unsigned int *result)
{
	//thread index
	int id = threadIdx.x + blockIdx.x * N;
	/* Copy state to local memory for efficiency */
	unsigned y = result[id];

	//Xorshift
	y = y ^ (y << 11);
	y = y ^ (y >> 7);
	y = y ^ (y >> 12);

	/* Copy state back to global memory */
	result[id] = y;
}

int main(int argc, char *argv[])
{
	int i;
	unsigned int total;
	unsigned int *devResults, *hostResults;

	/* Allocate space for results on host */
	hostResults = (unsigned int *)calloc(N * N, sizeof(int));

	/* Allocate space for results on device */
	CUDA_CALL(hipMalloc((void **)&devResults, N * N *
		sizeof(unsigned int)));

	/* Set results and seed to 0 */
	CUDA_CALL(hipMemset(devResults, 0, N * N *
		sizeof(unsigned int)));

	/* Setup prng states */
	unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
	mt19937 rand_num(seed);
	
	unsigned x = 123456789, y = 362436069, z = 521288629,
		w = 88675123, v = 5783321, d = 6615241;
	unsigned t;
	for (int i = 0; i < N * N; i++) {
		t = (x^(x >> 2));
		x = y;
		y = z;
		z = w;
		w = v;
		v = (v^(v << 4))^(t^(t << 1)); 
		//hostResults[i] = (d += 362437) + v;
		hostResults[i] = rand_num();
		rand_num();
	}
	CUDA_CALL(hipMemcpy(devResults, hostResults, N * N *
		sizeof(unsigned int), hipMemcpyHostToDevice));

	/*open file*/
	ofstream outfile;
	outfile.open("rng-test.txt");

	/* Generate and use pseudo-random  */
	for (i = 0; i < 128; i++) {
		my_generateRandom << <N, N >> > (devResults);

		/* Copy device memory to host */
		CUDA_CALL(hipMemcpy(hostResults, devResults, N * N *
			sizeof(unsigned int), hipMemcpyDeviceToHost));
		for (int j = 0; j < N * N; j++) {
			bitset<32> t(hostResults[j]);
			outfile << t;
		}
	}

	outfile.close();

	/* Cleanup */
	CUDA_CALL(hipFree(devResults));
	free(hostResults);
	printf("^^^^ kernel_example PASSED\n");
	return EXIT_SUCCESS;
}